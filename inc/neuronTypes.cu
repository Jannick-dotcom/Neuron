#include "neuronTypes.hpp"

#ifdef useGPU
void* operator new(size_t size)
{
    void *temp;
    hipError_t ret = hipMallocManaged(&temp, size);
    if(ret != hipError_t::hipSuccess)
    {
        printf("Layer: Malloc failed with code %d\n", ret);
        // exit(1);
    }
    return temp;
}
void operator delete(void* ptr) noexcept
{
    hipError_t ret = hipFree(ptr);
    if(ret != hipError_t::hipSuccess)
    {
        printf("Layer: delete %p failed with code %d\n", ptr, ret);
        // exit(1);
    }
}
void operator delete[](void* ptr) noexcept
{
    hipError_t ret = hipFree(ptr);
    if(ret != hipError_t::hipSuccess)
    {
        printf("Layer: delete[] failed with code %d\n", ret);
        // exit(1);
    }
}
#endif